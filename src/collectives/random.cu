#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"



__global__ void setup_kernel(hiprandState *state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(1234, 0, 0, &state[id]);
}

__global__ void generate_uniform_kernel(hiprandState *state, float* result) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  float random_number;
  hiprandState localState = state[id];
  random_number = hiprand_uniform(&localState);
  state[id] = localState;
  result[id] = random_number;
}
